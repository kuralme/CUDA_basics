#include "hip/hip_runtime.h"
#include "cuda_tools.h"

// CUDA Kernel for vector addition
__global__ void vectorAdd(int *a, int *b, int *c, int size)
{
    // Calculate global thread ID
    int tid = (blockIdx.x * blockDim.x) + threadIdx.x;

    // Vector boundary guard
    if(tid < size)
    { // Each thread adds a single element
        c[tid] = a[tid] + b[tid];
    }
}

// Initialize vector
void vector_init(int* vec, int size)
{
    for(int i=0; i < size; i++)
        vec[i] = rand() % 100;
}
void fvector_init(float* vec, int size)
{
	for (int i = 0; i < size; i++) {
		vec[i] = (float)(rand() % 100);
	}
}

// Check vector add result
void verify_result(int *a, int *b, int *c, int n) {
  for (int i = 0; i < n; i++) {
    assert(c[i] == a[i] + b[i]);
  }
}
void verify_fresult(float *a, float *b, float *c, float factor, int n) {
  for (int i = 0; i < n; i++) {
	assert(c[i] == factor * a[i] + b[i]);
  }
}


int main()
{
    printf("Vector adding with CPU vs CUDA cores\n");

    // Vector size of 2^16 (65536)
    int n = 1 << 16;
    // Allocation size for vectors
    size_t bytes = sizeof(int) * n;

    // Host vector pointers
    int *hostA, *hostB, *hostC, *f;
    hostA = (int*)malloc(bytes);
    hostB = (int*)malloc(bytes);
    hostC = (int*)malloc(bytes);
    f = (int*)malloc(bytes);

    // Init vectors with random values 0 to 100
    vector_init(hostA, n);
    vector_init(hostB, n);
    
    // =============  CPU  ==============================
    auto begin = std::chrono::high_resolution_clock::now();
    for(int i=0; i < n; i++)
    {
        f[i] = hostA[i] + hostB[i];
    }
    auto duration = std::chrono::duration_cast<std::chrono::microseconds>
        (std::chrono::high_resolution_clock::now() - begin);
    printf("CPU loop took\t\t\t%ld[us]\n", duration.count());
    
    // =============  CUDA  ======================================
    // Device vector pointers
    int *cudaA, *cudaB, *cudaC;
    hipMalloc(&cudaA, bytes);
    hipMalloc(&cudaB, bytes);
    hipMalloc(&cudaC, bytes);

    hipMemcpy(cudaA, hostA, bytes, hipMemcpyHostToDevice);
    hipMemcpy(cudaB, hostB, bytes, hipMemcpyHostToDevice);

    // Threadblock and grid size
    // if num of threads is an integer, block is one dimentional
    int NUM_THREADS = 1 << 10;
    int NUM_BLOCKS = (n + NUM_THREADS - 1) / NUM_THREADS;

    // Launch kernel on default stream without shmem
    auto begin1 = std::chrono::high_resolution_clock::now();
    vectorAdd <<< NUM_BLOCKS, NUM_THREADS>>> (cudaA, cudaB, cudaC, n);
    hipMemcpy(hostC, cudaC, bytes, hipMemcpyDeviceToHost);

    auto duration1 = std::chrono::duration_cast<std::chrono::microseconds>
        (std::chrono::high_resolution_clock::now() - begin1);
    verify_result(hostA, hostB, hostC, n);
    printf("CUDA baseline took\t\t%ld[us]\n", duration1.count());

    // Free unified memory
    hipFree(cudaA);
    hipFree(cudaB);
    hipFree(cudaC);


    // ============= Unified memory prefetch CUDA  ======================================

    // Declare unified memory pointers
    int *uniX, *uniY, *uniZ;

    // Allocation memory for these pointers
    hipMallocManaged(&uniX, bytes);
    hipMallocManaged(&uniY, bytes);
    hipMallocManaged(&uniZ, bytes);
    
    // Get the device ID for prefetching calls
    int id = hipGetDevice(&id);

    // Set some hints about the data and do some prefetching
    hipMemAdvise(uniX, bytes, hipMemAdviseSetPreferredLocation, hipCpuDeviceId);
    hipMemAdvise(uniY, bytes, hipMemAdviseSetPreferredLocation, hipCpuDeviceId);
    hipMemPrefetchAsync(uniZ, bytes, id);

    // Initialize vectors
    vector_init(uniX, bytes);
    vector_init(uniY, bytes);
    
    // Pre-fetch 'a' and 'b' arrays to the specified device (GPU)
    hipMemAdvise(uniX, bytes, hipMemAdviseSetReadMostly, id);
    hipMemAdvise(uniY, bytes, hipMemAdviseSetReadMostly, id);
    hipMemPrefetchAsync(uniX, bytes, id);
    hipMemPrefetchAsync(uniY, bytes, id);
    
    // Threads per CTA (65536 threads per CTA)
    int BLOCK_SIZE = 1 << 10;
    int GRID_SIZE = (n + BLOCK_SIZE - 1) / BLOCK_SIZE; // CTAs per Grid

    // Call CUDA kernel
    auto begin2 = std::chrono::high_resolution_clock::now();
    vectorAdd<<<GRID_SIZE, BLOCK_SIZE>>>(uniX, uniY, uniZ, n);

    // Wait for all previous operations before using values
    // We need this because we don't get the implicit synchronization 
    // of hipMemcpy like in the original example
    hipDeviceSynchronize();

    // Prefetch to the host (CPU)
    hipMemPrefetchAsync(uniX, bytes, hipCpuDeviceId);
    hipMemPrefetchAsync(uniY, bytes, hipCpuDeviceId);
    hipMemPrefetchAsync(uniZ, bytes, hipCpuDeviceId);

    auto duration2 = std::chrono::duration_cast<std::chrono::microseconds>
        (std::chrono::high_resolution_clock::now() - begin2);
    verify_result(uniX, uniY, uniZ, n);
    printf("Prefetched unified memory took\t%ld[us]\n", duration2.count());

    // Free unified memory
    hipFree(uniX);
    hipFree(uniY);
    hipFree(uniZ);

    // ============= CUDA Cublas =================================================
    float *hostcublA, *hostcublB, *hostcublC, *cublA, *cublB;
    hostcublA = (float*)malloc(bytes);
    hostcublB = (float*)malloc(bytes);
    hostcublC = (float*)malloc(bytes);
    hipMalloc(&cublA, bytes);
	hipMalloc(&cublB, bytes);

	fvector_init(hostcublA, n);
	fvector_init(hostcublB, n);

	// Create and initialize a new context
	hipblasHandle_t handle;
	hipblasCreate(&handle);

	// Copy the vectors over to the device
	hipblasSetVector(n, sizeof(float), hostcublA, 1, cublA, 1);
	hipblasSetVector(n, sizeof(float), hostcublB, 1, cublB, 1);

	// Launch simple saxpy kernel (single precision a * x + y)
    // Function signature: handle, # elements n, A, increment, B, increment
    auto begin3 = std::chrono::high_resolution_clock::now();
	const float scale = 2.0f;
	hipblasSaxpy(handle, n, &scale, cublA, 1, cublB, 1);
	hipblasGetVector(n, sizeof(float), cublB, 1, hostcublC, 1);

    auto duration3 = std::chrono::duration_cast<std::chrono::microseconds>
        (std::chrono::high_resolution_clock::now() - begin3);
	verify_fresult(hostcublA, hostcublB, hostcublC, scale, n);
    printf("Cublas kernel took\t\t%ld[us]\n", duration3.count());

	// Clean up the created handle
	hipblasDestroy(handle);

    // Free memory
    hipFree(cublA);
    hipFree(cublB);
    return 0;
}